#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "cs43805351.h"

static const int ThreadsPerBlock = 512;
static const double Delta = 0.006;
static const double xMid = 0.232997;
static const double yMid = 0.550325;

static __global__ void fractal(const int width, const int frames, unsigned char* pic)    {
  int elements = frames * width * width;
  // compute frames
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  // frames
  if(idx < elements)
  {
    const int col = idx % width;
    const int row = (idx/width) % width;
    const int frame = idx/(width*width);

    const double delta = Delta * pow(0.985, frame);
    const double xMin = xMid - delta;
    const double yMin = yMid - delta;
    const double dw = 2.0 * delta / width;
  // rows
    const double cy = yMin + row * dw;
  // columns
    const double cx = xMin + col * dw;
    double x = cx;
    double y = cy;
    int depth = 256;
    double x2, y2;
        do {
          x2 = x * x;
          y2 = y * y;
          y = 2 * x * y + cy;
          x = x2 - y2 + cx;
          depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0));
        pic[frame * width * width + row * width + col] = (unsigned char)depth;
  }
}

static void CheckCuda(){
   hipError_t e;
   hipDeviceSynchronize();
   if(hipSuccess != (e = hipGetLastError()))
   {
      fprintf(stderr, "CUDA error %d: %s", e, hipGetErrorString(e));
      exit (-1);
   }
}

int main(int argc, char *argv[])
{
  printf("Fractal v1.8\n");

  // check command line   if (argc != 3) {fprintf(stderr, "USAGE: %s frame_width num_frames\n", argv[0]); exit(-1);}
  const int width = atoi(argv[1]);
  if (width < 10) {fprintf(stderr, "ERROR: frame_width must be at least 10\n"); exit(-1);}
  const int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "ERROR: num_frames must be at least 1\n"); exit(-1);}
  printf("frames: %d\n", frames);
  printf("width: %d\n", width);

  //allocate mem for pic on GPU
  unsigned char *d_pic;
  const int size = sizeof(unsigned char)*frames*width*width;
  hipMalloc((void **)&d_pic, size);

  // allocate picture array for CPU
  unsigned char* pic = new unsigned char[frames*width*width];

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // call function for GPU (device).
  fractal<<<((frames*width*width) + ThreadsPerBlock-1)/ThreadsPerBlock, ThreadsPerBlock>>>(width, frames, d_pic);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  //(10) call checkCuda
  CheckCuda();

  // write result to BMP files
  if ((width <= 256) && (frames <= 100)) {
    for (int frame = 0; frame < frames; frame++) {
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      writeBMP(width, width, &pic[frame * width * width], name);
    }
  }

  delete [] pic;
  hipFree(d_pic);
  return 0;
}